#include "hip/hip_runtime.h"
#include "Dummy.cuh"

#include <iostream>

__global__ void dummyKernel() {
  printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}


void launchDummyKernel(){
  int numDevices = 0;
  hipGetDeviceCount( &numDevices );

  printf("Inside launchDummyKernel(). Found %d devices\n", numDevices);
  
  if ( numDevices > 0 ) {
    printf("Attempting to launch!\n");
    dummyKernel<<<2,8>>>();
    hipDeviceSynchronize();
    printf("Finished launch!\n");
  }
}
